#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "MSD_params.h"
#include "MSD_Configuration.h"

#include "timer.h"
#include "MSD_GPU_kernels_shared.cu"
#include "MSD_GPU_kernels_final.cu"
#include "MSD_GPU_kernels_1d.cu"
#include "MSD_GPU_kernels_2d.cu"
#include "MSD_GPU_kernels_3d.cu"

//#define MSD_DEBUG
//#define MSD_DEBUG_BLOCKS

//----------------------------------------------------------------------------------------------
//--------------------------> Invisible from outside
void MSD_init(){
	//---------> Specific nVidia stuff
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
}

template<typename input_type>
int MSD_outlier_rejection(float *d_MSD, size_t *d_MSD_nElements, input_type *d_input, MSD_Configuration *MSD_conf){
	hipError_t CUDA_error;
	CUDA_error = hipGetLastError();
	if(CUDA_error != hipSuccess) return(3);
	
	GpuTimer timer;
	double MSD_time = 0;
	
	timer.Start();
	//-------- Timed --->
	MSD_init();
	
	dim3 partials_gridSize  = MSD_conf->get_partial_gridSize();
	dim3 partials_blockSize = MSD_conf->get_partial_blockSize();
	dim3 final_gridSize     = MSD_conf->get_final_gridSize();
	dim3 final_blockSize    = MSD_conf->get_final_blockSize();
	hipStream_t hip_stream    = MSD_conf->get_CUDA_stream();
	
	float *d_partial_MSD       = MSD_conf->get_pointer_partial_MSD();
	int *d_partial_nElements   = MSD_conf->get_pointer_partial_nElements();
	int3 nSteps = MSD_conf->get_nSteps();
	int nDim     = MSD_conf->get_nDim(); 
	size_t dim_x = MSD_conf->get_dim_x();
	size_t dim_y = MSD_conf->get_dim_y();
	size_t dim_z = MSD_conf->get_dim_z();
	int offset = MSD_conf->get_offset();
	float sigma_threshold = MSD_conf->get_sigma_threshold();
	int nBlocks_total = MSD_conf->get_nBlocks_total();
	
	if(nDim==1){
		call_MSD_GPU_calculate_partials_1d_and_minmax(partials_gridSize, partials_blockSize, 0, hip_stream, d_input, d_partial_MSD, d_partial_nElements, nSteps.x, dim_x, offset);
	}
	else if(nDim==2){
		call_MSD_GPU_calculate_partials_2d_and_minmax(partials_gridSize, partials_blockSize, 0, hip_stream, d_input, d_partial_MSD, d_partial_nElements, dim_x, dim_y, offset);
	}
	else if(nDim==3){
		call_MSD_GPU_calculate_partials_3d_and_minmax(partials_gridSize, partials_blockSize, 0, hip_stream, d_input, d_partial_MSD, d_partial_nElements, dim_x, dim_y, offset);
	}
	call_MSD_GPU_final_regular(final_gridSize, final_blockSize, 0, hip_stream, d_partial_MSD, d_partial_nElements, d_MSD, d_MSD_nElements, nBlocks_total);
	//-------- Timed ---<
	timer.Stop();
	MSD_time += timer.Elapsed();
	//printf("Initial step done in	\033[1;32m%g\033[0m ms.\n", MSD_time);
	hipStreamSynchronize(hip_stream);
	
	#ifdef MSD_DEBUG
	float h_MSD[MSD_RESULTS_SIZE];
	size_t h_MSD_elements;
	hipMemcpy( h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy( &h_MSD_elements, d_MSD_nElements, sizeof(size_t), hipMemcpyDeviceToHost);
	printf("d_MSD=[%f;%f]; d_MSD_nElements=%d\n", h_MSD[0], h_MSD[1], (size_t) h_MSD_elements);
	#endif
	
	#ifdef MSD_DEBUG_BLOCKS
	float4 *h_blocks;
	h_blocks = new float4[nBlocks_total];
	hipMemcpy( h_blocks, d_partial_MSD, nBlocks_total*sizeof(float4), hipMemcpyDeviceToHost);
	int nBlocks_x = partials_gridSize.x;
	int nBlocks_y = partials_gridSize.y;
	for(int x=0; x<nBlocks_x; x++){
		for(int y=0; y<nBlocks_y; y++){
			printf("[%f;%f;%f;%f] ", h_blocks[y*nBlocks_x + x].x, h_blocks[y*nBlocks_x + x].y, h_blocks[y*nBlocks_x + x].z, h_blocks[y*nBlocks_x + x].w);
		}
		printf("\n");
	}
	delete [] h_blocks;
	#endif
	
	//TODO Criteria should be in the MSD_Config
	for(int i=0; i<5; i++){
		timer.Start();
		//-------- Timed --->
		if(nDim==1){
			call_MSD_BLN_calculate_partials_1d_and_minmax_with_outlier_rejection(partials_gridSize, partials_blockSize, 0, hip_stream, d_input, d_partial_MSD, d_partial_nElements, d_MSD, nSteps.x, dim_x, offset, sigma_threshold);
		}
		else if(nDim==2){
			call_MSD_BLN_calculate_partials_2d_and_minmax_with_outlier_rejection(partials_gridSize, partials_blockSize, 0, hip_stream, d_input, d_partial_MSD, d_partial_nElements, d_MSD, dim_x, dim_y, offset, sigma_threshold);
		}
		else if(nDim==3){
			call_MSD_BLN_calculate_partials_3d_and_minmax_with_outlier_rejection(partials_gridSize, partials_blockSize, 0, hip_stream, d_input, d_partial_MSD, d_partial_nElements, d_MSD, dim_x, dim_y, offset, sigma_threshold);
		}
		call_MSD_GPU_final_nonregular(final_gridSize, final_blockSize, 0, hip_stream, d_partial_MSD, d_partial_nElements, d_MSD, d_MSD_nElements, nBlocks_total);
		//-------- Timed ---<
		timer.Stop();
		MSD_time += timer.Elapsed();
		hipStreamSynchronize(hip_stream);
		
		#ifdef MSD_DEBUG
		float h_MSD[MSD_RESULTS_SIZE];
		size_t h_MSD_elements;
		hipMemcpy( h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy( &h_MSD_elements, d_MSD_nElements, sizeof(size_t), hipMemcpyDeviceToHost);
		printf("d_MSD=[%f;%f]; d_MSD_nElements=%d\n", h_MSD[0], h_MSD[1], (size_t) h_MSD_elements);
		#endif
	}
	
	MSD_conf->MSD_time = MSD_time;

	CUDA_error = hipPeekAtLastError();
	if(CUDA_error != hipSuccess) return(4);
	else return(0);
}



template<typename input_type>
int MSD_normal(float *d_MSD, size_t *d_MSD_nElements, input_type *d_input, MSD_Configuration *MSD_conf){
	hipError_t CUDA_error;
	CUDA_error = hipGetLastError();
	if(CUDA_error != hipSuccess) return(3);
	
	GpuTimer timer;
	double MSD_time = 0;
	
	timer.Start();	
	//-------- Timed --->
	MSD_init();
	
	dim3 partials_gridSize  = MSD_conf->get_partial_gridSize();
	dim3 partials_blockSize = MSD_conf->get_partial_blockSize();
	dim3 final_gridSize    = MSD_conf->get_final_gridSize();
	dim3 final_blockSize   = MSD_conf->get_final_blockSize();
	hipStream_t hip_stream    = MSD_conf->get_CUDA_stream();
	
	float *d_partial_MSD       = MSD_conf->get_pointer_partial_MSD();
	int *d_partial_nElements   = MSD_conf->get_pointer_partial_nElements();
	int3 nSteps = MSD_conf->get_nSteps();
	int nDim     = MSD_conf->get_nDim(); 
	size_t dim_x = MSD_conf->get_dim_x();
	size_t dim_y = MSD_conf->get_dim_y();
	size_t dim_z = MSD_conf->get_dim_z();
	int offset = MSD_conf->get_offset();
	float sigma_threshold = MSD_conf->get_sigma_threshold();
	int nBlocks_total = MSD_conf->get_nBlocks_total();
	
	if(nDim==1){
		call_MSD_GPU_calculate_partials_1d(partials_gridSize, partials_blockSize, 0, hip_stream, d_input, d_partial_MSD, d_partial_nElements, nSteps.x, dim_x, offset);
	}
	else if(nDim==2){
		call_MSD_GPU_calculate_partials_2d(partials_gridSize, partials_blockSize, 0, hip_stream, d_input, d_partial_MSD, d_partial_nElements, dim_x, dim_y, offset);
	}
	else if(nDim==3){
		call_MSD_GPU_calculate_partials_3d(partials_gridSize, partials_blockSize, 0, hip_stream, d_input, d_partial_MSD, d_partial_nElements, dim_x, dim_y, offset);
	}
	//hipStreamSynchronize(hip_stream);
	call_MSD_GPU_final_regular(final_gridSize, final_blockSize, 0, hip_stream, d_partial_MSD, d_partial_nElements, d_MSD, d_MSD_nElements, nBlocks_total);
	//-------- Timed ---<
	timer.Stop();
	MSD_time += timer.Elapsed();
	hipStreamSynchronize(hip_stream);
	
	#ifdef MSD_DEBUG
	float h_MSD[MSD_RESULTS_SIZE];
	hipMemcpy( h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy( h_MSD, d_MSD, sizeof(float), hipMemcpyDeviceToHost);
	printf("d_MSD=[%f;%f];\n", h_MSD[0], h_MSD[1]);
	#endif
	
	MSD_conf->MSD_time = MSD_time;
	CUDA_error = hipPeekAtLastError();
	if(CUDA_error != hipSuccess) return(4);
	else return(0);
}

//----------------------------------------------------------------------------------------------




//----------------------------------------------------------------------------------------------
//--------------------------> Visible from outside

// higher dimension would be done by configuring MSD_config and this will then launch the kernels.
// Kernels must be capable of processing 1D, 1D batched in direction of primary coordinate, 1D batched in direction of secondary coordinate, 2D, 3D
// Kernels should be capable of windowing the results.
// HOW TO IMPLEMENT:
// 1D: this would require that kernels would go in all directions x,y,z so we could sum in the kernel any size we want.
// 1D batched: For this I need to modify the 'final' kernels so they can produce multiple outputs. Problem is kernels themselves...
// 1D batched transposed: I'm not sure.
// 2D: fine 
// 2D batched: multiple outputs for 'final' kernel
// 3D: fine
// 3D batched: multiple kernels for 'final' kernel
// Direct mean and stdev for small batched jobs
// TODO list: 
MSD_Error MSD_GetMeanStdev(float *d_MSD, size_t *d_MSD_nElements, float *d_input, MSD_Configuration &MSD_conf){
	MSD_Error MSD_error;
	if( !MSD_conf.MSD_ready() ) return(5);
	
	//--------> MSD
	if( MSD_conf.MSD_outlier_rejection() ){
		MSD_error = MSD_outlier_rejection(d_MSD, d_MSD_nElements, d_input, &MSD_conf);
	}
	else {
		MSD_error = MSD_normal(d_MSD, d_MSD_nElements, d_input, &MSD_conf);
	}
	
	return(MSD_error);
}

//-----------------------------------------------------------------------------------------<

