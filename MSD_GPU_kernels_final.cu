#include "hip/hip_runtime.h"
#ifndef MSD_GPU_FINAL_CU
#define MSD_GPU_FINAL_CU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "MSD_params.h"
#include "MSD_GPU_kernels_shared.cu"

//----------------------------------------------------------------------------------------
//--------> Kernels
template<typename nelements_accumulator>
__global__ void MSD_GPU_final_regular(float *d_partial_MSD, int *d_partial_nElements, float *d_output_MSD, nelements_accumulator *d_output_nElements, int size) {
	__shared__ float s_par_MSD[2*MSD_WARP*MSD_WARP];
	__shared__ nelements_accumulator s_par_nElements[MSD_WARP*MSD_WARP];

	float M, S;
	nelements_accumulator j;
	
	Sum_partials_regular( &M, &S, &j, &d_partial_MSD[blockIdx.x*size*MSD_PARTIAL_SIZE], &d_partial_nElements[blockIdx.x*size], s_par_MSD, s_par_nElements, size);

	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_output_MSD[MSD_RESULTS_SIZE*blockIdx.x] = M / (double) j;
		d_output_MSD[MSD_RESULTS_SIZE*blockIdx.x + 1] = sqrt(S / (double) j);
		d_output_nElements[blockIdx.x] = j;
	}
}


template<typename nelements_accumulator>
__global__ void MSD_GPU_final_regular(float *d_partial_MSD, int *d_partial_nElements, float *d_output_MSD, nelements_accumulator *d_output_nElements, float *d_previous_MSD, nelements_accumulator *d_previous_nElements, int size) {
	__shared__ float s_par_MSD[2*MSD_WARP*MSD_WARP];
	__shared__ nelements_accumulator s_par_nElements[MSD_WARP*MSD_WARP];

	float M, S;
	nelements_accumulator j;
	
	Sum_partials_regular( &M, &S, &j, &d_partial_MSD[blockIdx.x*size*MSD_PARTIAL_SIZE], &d_partial_nElements[blockIdx.x*size], s_par_MSD, s_par_nElements, size);

	if((*d_previous_nElements)>0){
		Merge(&M, &S, &j, d_previous_MSD[0], d_previous_MSD[1], (*d_previous_nElements));
	}
	
	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_output_MSD[MSD_RESULTS_SIZE*blockIdx.x] = M / (double) j;
		d_output_MSD[MSD_RESULTS_SIZE*blockIdx.x + 1] = sqrt(S / (double) j);
		d_output_nElements[blockIdx.x] = j;
		d_previous_MSD[MSD_RESULTS_SIZE*blockIdx.x] = M;
		d_previous_MSD[MSD_RESULTS_SIZE*blockIdx.x + 1] = S;
		d_previous_nElements[blockIdx.x] = j;
	}
}


template<typename nelements_accumulator>
__global__ void MSD_GPU_final_nonregular(float *d_partial_MSD, int *d_partial_nElements, float *d_output_MSD, nelements_accumulator *d_output_nElements, int size) {
	__shared__ float s_par_MSD[2*MSD_WARP*MSD_WARP];
	__shared__ nelements_accumulator s_par_nElements[MSD_WARP*MSD_WARP];
	
	float M, S;
	nelements_accumulator j;

	Sum_partials_nonregular( &M, &S, &j, &d_partial_MSD[blockIdx.x*size*MSD_PARTIAL_SIZE], &d_partial_nElements[blockIdx.x*size], s_par_MSD, s_par_nElements, size);
	
	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_output_MSD[MSD_RESULTS_SIZE*blockIdx.x]     = M / (double) j;
		d_output_MSD[MSD_RESULTS_SIZE*blockIdx.x + 1] = sqrt(S / (double) j);
		d_output_nElements[blockIdx.x] = j;
		//printf("Mean=%f; Stdev=%f; j=%e;\n", d_output_MSD[0], d_output_MSD[1], (double) j);
	}
}


template<typename nelements_accumulator>
__global__ void MSD_GPU_final_nonregular(float *d_partial_MSD, int *d_partial_nElements, float *d_output_MSD, nelements_accumulator *d_output_nElements, float *d_previous_MSD, nelements_accumulator *d_previous_nElements, int size) {
	__shared__ float s_par_MSD[2*MSD_WARP*MSD_WARP];
	__shared__ nelements_accumulator s_par_nElements[MSD_WARP*MSD_WARP];
	
	float M, S;
	nelements_accumulator j;

	Sum_partials_nonregular( &M, &S, &j, &d_partial_MSD[blockIdx.x*size*MSD_PARTIAL_SIZE], &d_partial_nElements[blockIdx.x*size], s_par_MSD, s_par_nElements, size);
	
	if((*d_previous_nElements)>0){
		Merge(&M, &S, &j, d_previous_MSD[0], d_previous_MSD[1], (*d_previous_nElements));
	}
	
	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_output_MSD[MSD_RESULTS_SIZE*blockIdx.x] = M / (double) j;
		d_output_MSD[MSD_RESULTS_SIZE*blockIdx.x + 1] = sqrt(S / (double) j);
		d_output_nElements[blockIdx.x] = j;
		d_previous_MSD[MSD_RESULTS_SIZE*blockIdx.x] = M;
		d_previous_MSD[MSD_RESULTS_SIZE*blockIdx.x + 1] = S;
		d_previous_nElements[blockIdx.x] = j;
	}
}

//----------------------------------------------------------------------------------------<



//----------------------------------------------------------------------------
//---------------> Simple C wrappers
// NOTE: for the moment this will work only for fp32
void call_MSD_GPU_final_regular(const dim3 &grid_size, const dim3 &block_size, int shared_memory_bytes, hipStream_t streams, float *d_partial_MSD, int *d_partial_nElements, float *d_output_MSD, size_t *d_output_nElements, int size){
	MSD_GPU_final_regular<<< grid_size, block_size, shared_memory_bytes, streams>>>(d_partial_MSD, d_partial_nElements, d_output_MSD, d_output_nElements, size);
}

void call_MSD_GPU_final_regular(const dim3 &grid_size, const dim3 &block_size, int shared_memory_bytes, hipStream_t streams, float *d_partial_MSD, int *d_partial_nElements, float *d_output_MSD, size_t *d_output_nElements, float *d_previous_MSD, size_t *d_previous_nElements, int size){
	MSD_GPU_final_regular<<< grid_size, block_size, shared_memory_bytes, streams>>>(d_partial_MSD, d_partial_nElements, d_output_MSD, d_output_nElements, d_previous_MSD, d_previous_nElements, size);
}

void call_MSD_GPU_final_nonregular(const dim3 &grid_size, const dim3 &block_size, int shared_memory_bytes, hipStream_t streams, float *d_partial_MSD, int *d_partial_nElements, float *d_output_MSD, size_t *d_output_nElements, int size){
	MSD_GPU_final_nonregular<<< grid_size, block_size, shared_memory_bytes, streams>>>(d_partial_MSD, d_partial_nElements, d_output_MSD, d_output_nElements, size);
}

void call_MSD_GPU_final_nonregular(const dim3 &grid_size, const dim3 &block_size, int shared_memory_bytes, hipStream_t streams, float *d_partial_MSD, int *d_partial_nElements, float *d_output_MSD, size_t *d_output_nElements, float *d_previous_MSD, size_t *d_previous_nElements, int size){
	MSD_GPU_final_nonregular<<< grid_size, block_size, shared_memory_bytes, streams>>>(d_partial_MSD, d_partial_nElements, d_output_MSD, d_output_nElements, d_previous_MSD, d_previous_nElements, size);
}

//----------------------------------------------------------------------------<

#endif
