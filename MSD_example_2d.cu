#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <vector>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "timer.h"
#include "MSD_GPU_library.h"

float max_error = 1.0e-4;

void Generate_dataset(float *h_input, size_t dim_x, size_t dim_y, size_t offset, int nBatches, float scale, float spike_ratio){
	for(size_t b=0; b<(size_t) nBatches; b++){
		for(size_t y=0; y<dim_y; y++){
			for(size_t x=0; x<dim_x; x++){
				size_t pos = b*dim_x*dim_y + y*dim_x + x;
				h_input[pos] = ( rand() / ((float) RAND_MAX) ) * ((float) (b+1)) * scale;
				if(x>(dim_x-offset)) h_input[pos] = 10000;
			}
		}
		
		long int nSpikes = ( ((float) dim_y)*((float) dim_x))*spike_ratio;
		for(long int f=0; f<nSpikes; f++){
			size_t x = (size_t) (((double) dim_x)*((double) rand() / (double) RAND_MAX));
			size_t y = (size_t) (((double) dim_y)*((double) rand() / (double) RAND_MAX));
			if(x<dim_x && y<dim_y){
				size_t pos = b*dim_x*dim_y + y*dim_x + x;
				h_input[pos] = 2.0*((float) (b+1))*scale;
			}
		}
	} // batches
}

void Generate_dataset_for_offset_test(float *h_input, size_t dim_x, size_t dim_y, size_t offset){
	for(size_t y=0; y<dim_y; y++){
		for(size_t x=0; x<dim_x; x++){
			size_t pos = y*dim_x + x;
			h_input[pos] = ( rand() / ((float) RAND_MAX) ) * (10000.0/((double) dim_x))*((double) x/((double) dim_x));
			if(x>(dim_x-offset)) h_input[pos] = 10000;
		}
	}
}

//---------------------------------------------------------------------------------
//-------> Kahan MSD
void d_kahan_summation(float *signal, size_t dim_y, size_t dim_x, size_t offset, float *result, float *error){
	double sum;
	double sum_error;
	double a,b;
	
	sum=0;
	sum_error=0;
	for(size_t d=0;d<dim_y; d++){
		for(size_t s=0; s<(dim_x-offset); s++){
			a=signal[(size_t) (d*dim_x + s)]-sum_error;
			b=sum+a;
			sum_error=(b-sum);
			sum_error=sum_error-a;
			sum=b;
		}
	}
	*result=sum;
	*error=sum_error;
}

void d_kahan_sd(float *signal, size_t dim_y, size_t dim_x, size_t offset, double mean, float *result, float *error){
	double sum;
	double sum_error;
	double a,b,dtemp;
	
	sum=0;
	sum_error=0;
	for(size_t d=0;d<dim_y; d++){
		for(size_t s=0; s<(dim_x-offset); s++){
			dtemp=(signal[(size_t) (d*dim_x + s)]-sum_error - mean);
			a=dtemp*dtemp;
			b=sum+a;
			sum_error=(b-sum);
			sum_error=sum_error-a;
			sum=b;
		}
	}
	*result=sum;
	*error=sum_error;
}

void MSD_Kahan(float *h_input, size_t dim_y, size_t dim_x, size_t offset, double *mean, double *sd){
	float error, signal_mean, signal_sd;
	size_t nElements=dim_y*(dim_x-offset);
	
	d_kahan_summation(h_input, dim_y, dim_x, offset, &signal_mean, &error);
	signal_mean=signal_mean/nElements;
	
	d_kahan_sd(h_input, dim_y, dim_x, offset, signal_mean, &signal_sd, &error);
	signal_sd=sqrt(signal_sd/nElements);

	*mean=signal_mean;
	*sd=signal_sd;
}
//-------> Kahan MSD
//---------------------------------------------------------------------------------

bool Check_memory(size_t dim_x, size_t dim_y){
	size_t free_memory, total_memory, required_memory;
	hipMemGetInfo(&free_memory,&total_memory);
	required_memory = dim_x*dim_y*sizeof(float) + 1 + MSD_RESULTS_SIZE;
	if(required_memory>free_memory) {
		printf("Device has %0.3f MB of total memory, which %0.3f MB is available. Memory required %0.3f MB\n", (float) total_memory/(1024.0*1024.0), (float) free_memory/(1024.0*1024.0) , (float) required_memory/(1024.0*1024.0));
		printf("\n \n Array is too big for the device! \n \n"); 
		return(1);
	}
	else return(0);
}

int MSD(float *h_input, size_t dim_x, size_t dim_y, size_t offset, int nBatches, bool outlier_rejection, float outlier_rejection_sigma, MSD_Error *error, int verbose = 0){
	GpuTimer timer;
	
	int MSD_size = MSD_RESULTS_SIZE*nBatches*sizeof(float);
	int MSD_elements_size = nBatches*sizeof(size_t);
	size_t input_size = dim_x*dim_y*nBatches*sizeof(float);
	
	//---------> CPU Memory allocation
	float *h_MSD;
	size_t *h_MSD_nElements;
	h_MSD 		    = (float *)malloc(MSD_size);
	h_MSD_nElements = (size_t *)malloc(MSD_elements_size);
	memset(h_MSD, 0.0, MSD_size);
	memset(h_MSD_nElements, 0.0, MSD_elements_size);
	
	//---------> GPU Memory allocation
	float *d_input;
	float *d_MSD;
	size_t *d_MSD_nElements;
	if ( hipSuccess != hipMalloc((void **) &d_input, input_size)) {
		printf("CUDA API error while allocating GPU memory\n");
	}
	if ( hipSuccess != hipMalloc((void **) &d_MSD, MSD_size)) {
		printf("CUDA API error while allocating GPU memory\n");
	}
	if ( hipSuccess != hipMalloc((void **) &d_MSD_nElements, MSD_elements_size)) {
		printf("CUDA API error while allocating GPU memory\n");
	}
	
	//---------> Copy data to the device
	hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
	
	//---------> Create MSD plan
	MSD_Error MSD_error;
	MSD_Configuration MSD_conf;
	std::vector<size_t> dimensions={dim_y,dim_x}; // dimensions of the data. Fastest moving coordinate is at the end.
	MSD_error = MSD_conf.Create_MSD_Plan(dimensions, offset, outlier_rejection, outlier_rejection_sigma, nBatches);
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	*error = MSD_error;
	
	//---------> Get mean and stdev through library
	timer.Start();
	MSD_error = MSD_GetMeanStdev(d_MSD, d_MSD_nElements, d_input, MSD_conf);
	timer.Stop();
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	if(verbose) printf("Calculation of mean and standard deviation took %g ms\n", timer.Elapsed());
	
	//MSD GPU library outputs one float array (for example d_MSD) which contains mean as d_MSD[0] and standard deviation as d_MSD[1]
	
	//---------> Copy data to the host
	hipMemcpy( h_MSD, d_MSD, MSD_size, hipMemcpyDeviceToHost);
	hipMemcpy( h_MSD_nElements, d_MSD_nElements, MSD_elements_size, hipMemcpyDeviceToHost);
	
	//---> Checks
	int no_check_error = 1;
	for(int b=0; b<nBatches; b++){
		double signal_mean, signal_sd, merror, sderror;
		MSD_Kahan(&h_input[b*dim_x*dim_y], dim_y, dim_x, offset, &signal_mean, &signal_sd);
		merror  = sqrt((signal_mean-h_MSD[b*MSD_RESULTS_SIZE])*(signal_mean-h_MSD[b*MSD_RESULTS_SIZE]));
		sderror = sqrt((signal_sd-h_MSD[b*MSD_RESULTS_SIZE + 1])*(signal_sd-h_MSD[b*MSD_RESULTS_SIZE + 1]));
		if(merror>max_error && sderror>max_error) no_check_error = no_check_error*0;
	
		if(verbose) {
			printf("GPU results: Mean: %e, Standard deviation: %e; Number of elements:%zu;\n", h_MSD[b*MSD_RESULTS_SIZE], h_MSD[b*MSD_RESULTS_SIZE + 1], h_MSD_nElements[b]);
			printf("CPU results: Mean: %e, Standard deviation: %e;\n",signal_mean, signal_sd);
			if(!outlier_rejection) printf("Difference CPU-GPU Mean:%e; Standard deviation:%e;\n", merror, sderror);
		}
	}
	
	free(h_MSD);
	free(h_MSD_nElements);
	if ( hipSuccess != hipFree(d_input)) {
		printf("CUDA API error while deallocating GPU memory\n");
	}
	if ( hipSuccess != hipFree(d_MSD)) {
		printf("CUDA API error while deallocating GPU memory\n");
	}
	if ( hipSuccess != hipFree(d_MSD_nElements)) {
		printf("CUDA API error while deallocating GPU memory\n");
	}
	MSD_error = MSD_conf.Destroy_MSD_Plan();
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	
	if(no_check_error==1) return(1);
	else return(0);
}


int main(int argc, char* argv[]) {
	size_t dim_y;
	size_t dim_x;
	size_t offset;
	int device_id;

	// Check!
	char * pEnd;
	if (argc==5) {
		dim_y     = strtol(argv[1],&pEnd,10);
		dim_x     = strtol(argv[2],&pEnd,10);
		offset    = strtol(argv[3],&pEnd,10);
		device_id = strtol(argv[4],&pEnd,10);
	}
	else {
		printf("Argument error!\n");
		printf(" 1) dimensions x\n");
		printf(" 2) dimensions y\n");
		printf(" 3) offset\n");
		printf(" 4) device id\n");
		printf("Example: MSD_example_2d.exe 1000000 1500 15 0\n");
        return(1);
	}
	
	size_t input_size = dim_x*dim_y;

	//---------> Device initialization
	int deviceCount;
	hipError_t error_id;
	error_id = hipGetDeviceCount(&deviceCount);
	if(error_id != hipSuccess) {
		printf("CUDA ERROR: %s\n", hipGetErrorString(error_id) );
		return(1);
	}
	if(device_id>=deviceCount) {
		printf("Selected device is not available! Device id is %d;\n", device_id);
		return(1);
	}
	if (hipSetDevice(device_id) != hipSuccess) {
		printf("ERROR! unable to set the device with id %d.\n", device_id);
		return(1);
	}
	
	size_t free_memory, total_memory;
	hipMemGetInfo(&free_memory,&total_memory);
	offset = 0;
	if(Check_memory(dim_x, dim_y)) return(1);
	
	//---------> CPU Memory allocation
	float *h_input;
	srand(time(NULL));
	MSD_Error MSD_error;
	
	//============================== Mean and standard deviation ==========================o
	printf("Allocating host memory\n");
	h_input = (float *)malloc(input_size*sizeof(float));
	printf("Generating data\n");
	Generate_dataset(h_input, dim_x, dim_y, offset, 1, 1.0, 0.05);

	printf("\nMean and standard deviation:\n"); 
	MSD(h_input, dim_x, dim_y, offset, 1, false, 0.0, &MSD_error, 1);
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	
	printf("\nMean and standard deviation with outlier rejection:\n"); 
	MSD(h_input, dim_x, dim_y, offset, 1, true, 3.0, &MSD_error, 1);
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	
	printf("\nBatched mean and standard deviation:\n"); 
	int nBatches = 16;
	dim_x = dim_x/4;
	dim_y = dim_y/4;
	Generate_dataset(h_input, dim_x, dim_y, offset, nBatches, 1.0, 0.05);
	MSD(h_input, dim_x, dim_y, offset, nBatches, false, 0.0, &MSD_error, 1);
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	free(h_input);
	//--------------------<
	
	//============================== Unit tests ==========================o
	printf("\n\n====================================\n");
	printf("Unit tests:\n");
	int test_passed = 1;
	
	printf("Different data size:");
	test_passed = 1;
	int sizes_size = 9;
	size_t sizes[] = {2, 3, 7, 50, 1000, 10000, (size_t) sqrt((free_memory/(sizeof(float)*4))), (size_t) sqrt((free_memory/(sizeof(float)*2))), (size_t) sqrt(((free_memory/sizeof(float))*(3.0/4.0)))};
	for(int f=0; f<sizes_size; f++){
		offset = 0;
		dim_x = sizes[f];
		dim_y = sizes[f];
		if(!Check_memory(dim_x,dim_y)) {
		h_input = (float *)malloc(dim_x*dim_y*sizeof(float));
		Generate_dataset(h_input, dim_x, dim_y, offset, 1, 1.0, 0.05);
		int error = MSD(h_input, dim_x, dim_y, offset, 1, false, 0.0, &MSD_error, 0);
		test_passed = test_passed*error;
		free(h_input);
		}
		printf(".");
		fflush(stdout);
	}
	if(test_passed) printf("PASSED\n");
	else printf("FAILED\n");
	printf("\n");	
	
	printf("Different offset values:");
	test_passed = 1;
	dim_x = 10000;
	dim_y = 10000;
	std::vector<size_t> offs{0, dim_x/4, dim_x/2, (size_t) ((3.0/4.0)*dim_x)};
	for(size_t f=0; f<offs.size(); f++){
		offset = offs[f];
		h_input = (float *)malloc(dim_x*dim_y*sizeof(float));
		Generate_dataset_for_offset_test(h_input, dim_x, dim_y, offset);
		int error = MSD(h_input, dim_x, dim_y, offset, 1, false, 0.0, &MSD_error, 0);
		test_passed = test_passed*error;
		free(h_input);
		printf(".");
		fflush(stdout);
	}
	if(test_passed) printf("PASSED\n");
	else printf("FAILED\n");
	printf("\n");
	
	
	
	//----------------------------------------------------------->
	printf("Check individual blocks: ");
	{
		dim_x = 10000;
		dim_y = 10000;
		offset = 0;
		
		MSD_Error MSD_error;
		MSD_Configuration MSD_conf;
		std::vector<size_t> dimensions={dim_y, dim_x};
		MSD_error = MSD_conf.Create_MSD_Plan(dimensions, offset, false, 0.0, 1);
		if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
		
		dim3 partial_gridsize = MSD_conf.get_partial_gridSize();
		int3 nSteps = MSD_conf.get_nSteps();
		int nBlocks = partial_gridsize.x*partial_gridsize.y;
		int block_size_x = MSD_NTHREADS;
		int block_size_y = nSteps.y;
		size_t partial_MSD_size = nBlocks*MSD_PARTIAL_SIZE*sizeof(float);
		size_t partial_MSD_nElements_size = nBlocks*sizeof(int);
		size_t MSD_size = MSD_RESULTS_SIZE*sizeof(float);
		size_t MSD_elements_size = sizeof(size_t);
		size_t input_size = dim_x*dim_y*sizeof(float);
		float *h_MSD;
		size_t *h_MSD_nElements;
		float *h_partial_MSD;
		int *h_partial_MSD_nElements;
		h_input         = (float *)malloc(input_size);
		h_MSD 		    = (float *)malloc(MSD_size);
		h_MSD_nElements = (size_t *)malloc(MSD_elements_size);
		h_partial_MSD   = (float *)malloc(partial_MSD_size);
		h_partial_MSD_nElements = (int *)malloc(partial_MSD_nElements_size);
		memset(h_MSD, 0.0, MSD_size);
		memset(h_MSD_nElements, 0.0, MSD_elements_size);
		
		float *d_input;
		float *d_MSD;
		size_t *d_MSD_nElements;
		if ( hipSuccess != hipMalloc((void **) &d_input, input_size)) {
			printf("CUDA API error while allocating GPU memory\n");
		}
		if ( hipSuccess != hipMalloc((void **) &d_MSD, MSD_size)) {
			printf("CUDA API error while allocating GPU memory\n");
		}
		if ( hipSuccess != hipMalloc((void **) &d_MSD_nElements, MSD_elements_size)) {
			printf("CUDA API error while allocating GPU memory\n");
		}
		
		
		Generate_dataset(h_input, dim_x, dim_y, offset, 1, 1.0, 0.05);
		hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
		MSD_error = MSD_GetMeanStdev(d_MSD, d_MSD_nElements, d_input, MSD_conf);
		hipMemcpy( h_MSD, d_MSD, MSD_size, hipMemcpyDeviceToHost);
		hipMemcpy( h_MSD_nElements, d_MSD_nElements, MSD_elements_size, hipMemcpyDeviceToHost);
		hipMemcpy( h_partial_MSD, MSD_conf.get_pointer_partial_MSD(), partial_MSD_size, hipMemcpyDeviceToHost);
		hipMemcpy( h_partial_MSD_nElements, MSD_conf.get_pointer_partial_nElements(), partial_MSD_nElements_size, hipMemcpyDeviceToHost);

		int no_check_error = 1;
		
		int nBlocks_x = (int) ((dim_x + block_size_x - 1)/block_size_x);
		int nBlocks_y = (int) ((dim_y + block_size_y - 1)/block_size_y);
		float *tempdata;
		for(int by = 0; by<nBlocks_y; by++){
			for(int bx = 0; bx<nBlocks_x; bx++){
				double signal_mean, signal_sd, merror, sderror;
				int limit_x = (bx==(nBlocks_x-1)?(dim_x-bx*block_size_x):block_size_x);
				int limit_y = (by==(nBlocks_y-1)?(dim_y-by*block_size_y):block_size_y);
				tempdata = new float[limit_x*limit_y];
				for(int ty = 0; ty<limit_y; ty++){
					for(int tx = 0; tx<limit_x; tx++){
						size_t pos = (by*block_size_y + ty)*dim_x + bx*block_size_x + tx;
						tempdata[ty*limit_x + tx]=h_input[pos];
					}
				}
				MSD_Kahan(tempdata, limit_y, limit_x, 0, &signal_mean, &signal_sd);
				size_t respos = by*nBlocks_x + bx;
				float GPU_mean = h_partial_MSD[MSD_PARTIAL_SIZE*respos]/((double) h_partial_MSD_nElements[respos]);
				float GPU_sd = sqrt(h_partial_MSD[MSD_PARTIAL_SIZE*respos + 1]/((double) h_partial_MSD_nElements[respos]));
				merror  = sqrt((signal_mean-GPU_mean)*(signal_mean-GPU_mean));
				sderror = sqrt((signal_sd-GPU_sd)*(signal_sd-GPU_sd));
				if(merror>max_error && sderror>max_error) no_check_error = no_check_error*0;
				delete [] tempdata;
			}
		}
		
		
		if ( hipSuccess != hipFree(d_input)) {
			printf("CUDA API error while deallocating GPU memory\n");
		}
		if ( hipSuccess != hipFree(d_MSD)) {
			printf("CUDA API error while deallocating GPU memory\n");
		}
		if ( hipSuccess != hipFree(d_MSD_nElements)) {
			printf("CUDA API error while deallocating GPU memory\n");
		}
		
		free(h_input);
		free(h_MSD);
		free(h_MSD_nElements);
		free(h_partial_MSD);
		free(h_partial_MSD_nElements);
		
		if(no_check_error==1) printf("PASSED\n");
		else printf("FAILED\n");
	}
	
	
	
	//----------------------------------------------------------->
	printf("\nReuse of the MSD plan: ");
	{
		dim_x = 10000;
		dim_y = 10000;
		offset = 0;
		size_t MSD_size = MSD_RESULTS_SIZE*sizeof(float);
		size_t MSD_elements_size = sizeof(size_t);
		size_t input_size = dim_x*dim_y*sizeof(float);
		float *h_MSD;
		size_t *h_MSD_nElements;
		h_input         = (float *)malloc(input_size);
		h_MSD 		    = (float *)malloc(MSD_size);
		h_MSD_nElements = (size_t *)malloc(MSD_elements_size);
		memset(h_MSD, 0.0, MSD_size);
		memset(h_MSD_nElements, 0.0, MSD_elements_size);
		float *d_input;
		float *d_MSD;
		size_t *d_MSD_nElements;
		if ( hipSuccess != hipMalloc((void **) &d_input, input_size)) {
			printf("CUDA API error while allocating GPU memory\n");
		}
		if ( hipSuccess != hipMalloc((void **) &d_MSD, MSD_size)) {
			printf("CUDA API error while allocating GPU memory\n");
		}
		if ( hipSuccess != hipMalloc((void **) &d_MSD_nElements, MSD_elements_size)) {
			printf("CUDA API error while allocating GPU memory\n");
		}
		
		MSD_Error MSD_error;
		MSD_Configuration MSD_conf;
		std::vector<size_t> dimensions={dim_y, dim_x};
		MSD_error = MSD_conf.Create_MSD_Plan(dimensions, offset, false, 0.0, 1);
		if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
		
		int no_check_error = 1;
		for(int f=0; f<10; f++){
			Generate_dataset(h_input, dim_x, dim_y, offset, 1, (float) f+1.0, 0.05);
			hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
			MSD_error = MSD_GetMeanStdev(d_MSD, d_MSD_nElements, d_input, MSD_conf);
			hipMemcpy( h_MSD, d_MSD, MSD_size, hipMemcpyDeviceToHost);
			hipMemcpy( h_MSD_nElements, d_MSD_nElements, MSD_elements_size, hipMemcpyDeviceToHost);
			
			//--- check
			double signal_mean, signal_sd, merror, sderror;
			MSD_Kahan(h_input, dim_y, dim_x, offset, &signal_mean, &signal_sd);
			merror  = sqrt((signal_mean-h_MSD[0])*(signal_mean-h_MSD[0]));
			sderror = sqrt((signal_sd-h_MSD[1])*(signal_sd-h_MSD[1]));
			if(merror>max_error && sderror>max_error) no_check_error = no_check_error*0;
		}
		
		MSD_error = MSD_conf.Destroy_MSD_Plan();
		free(h_input);
		free(h_MSD);
		free(h_MSD_nElements);
		if ( hipSuccess != hipFree(d_input)) {
			printf("CUDA API error while deallocating GPU memory\n");
		}
		if ( hipSuccess != hipFree(d_MSD)) {
			printf("CUDA API error while deallocating GPU memory\n");
		}
		if ( hipSuccess != hipFree(d_MSD_nElements)) {
			printf("CUDA API error while deallocating GPU memory\n");
		}
		
		if(no_check_error==1) printf("PASSED\n");
		else printf("FAILED\n");
	}
	
	
	printf("x-dimension=0 : ");
	{
		int no_check_error=1;
		MSD_Error MSD_error;
		MSD_Configuration MSD_conf;
		std::vector<size_t> dimensions={1000, 0};
		MSD_error = MSD_conf.Create_MSD_Plan(dimensions, offset, false, 0.0, 1);
		if(MSD_error!=10) no_check_error = no_check_error*0;
		
		if(no_check_error==1) printf("PASSED\n");
		else printf("FAILED\n");
	}
	
	printf("y-dimension=0 : ");
	{
		int no_check_error=1;
		MSD_Error MSD_error;
		MSD_Configuration MSD_conf;
		std::vector<size_t> dimensions={0, 1000};
		MSD_error = MSD_conf.Create_MSD_Plan(dimensions, offset, false, 0.0, 1);
		if(MSD_error!=11) no_check_error = no_check_error*0;
		
		if(no_check_error==1) printf("PASSED\n");
		else printf("FAILED\n");
	}



	return (0);
}
