#include "hip/hip_runtime.h"
#include "debug.h"

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <vector>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "timer.h"
#include "MSD_GPU_library.h"

void Generate_dataset(float *h_input, size_t dim_x, size_t dim_y, size_t dim_z, int offset){
	for(size_t z=0; z<dim_z; z++){
		for(size_t y=0; y<dim_y; y++){
			for(size_t x=0; x<dim_x; x++){
				h_input[z*dim_x*dim_y + y*dim_x + x]=rand() / (float)RAND_MAX;
				if(x>(dim_x-offset)) h_input[z*dim_x*dim_y + y*dim_x + x] = 10000;
			}
		}
	}
}

//---------------------------------------------------------------------------------
//-------> Kahan MSD
void d_kahan_summation(float *signal, size_t dim_x, size_t dim_y, size_t dim_z, size_t offset, float *result, float *error){
	double sum;
	double sum_error;
	double a,b;
	
	sum=0;
	sum_error=0;
	for(size_t z=0; z<dim_z; z++){
		for(size_t y=0; y<dim_y; y++){
			for(size_t x=0; x<(dim_x-offset); x++){
				a=signal[(size_t) (z*dim_x*dim_y + y*dim_x + x)]-sum_error;
				b=sum+a;
				sum_error=(b-sum);
				sum_error=sum_error-a;
				sum=b;
			}
		}
	}
	*result=sum;
	*error=sum_error;
}

void d_kahan_sd(float *signal, size_t dim_x, size_t dim_y, size_t dim_z, size_t offset, double mean, float *result, float *error){
	double sum;
	double sum_error;
	double a,b,dtemp;
	
	sum=0;
	sum_error=0;
	for(size_t z=0; z<dim_z; z++){
		for(size_t y=0; y<dim_y; y++){
			for(size_t x=0; x<(dim_x-offset); x++){
				dtemp=(signal[(size_t) (z*dim_x*dim_y + y*dim_x + x)]-sum_error - mean);
				a=dtemp*dtemp;
				b=sum+a;
				sum_error=(b-sum);
				sum_error=sum_error-a;
				sum=b;
			}
		}
	}
	*result=sum;
	*error=sum_error;
}

void MSD_Kahan(float *h_input, size_t dim_x, size_t dim_y, size_t dim_z, size_t offset, double *mean, double *sd){
	float error, signal_mean, signal_sd;
	size_t nElements=dim_z*dim_y*(dim_x-offset);
	
	d_kahan_summation(h_input, dim_x, dim_y, dim_z, offset, &signal_mean, &error);
	signal_mean=signal_mean/nElements;
	
	d_kahan_sd(h_input, dim_x, dim_y, dim_z, offset, signal_mean, &signal_sd, &error);
	signal_sd=sqrt(signal_sd/nElements);

	*mean=signal_mean;
	*sd=signal_sd;
}
//-------> Kahan MSD
//---------------------------------------------------------------------------------


int main(int argc, char* argv[]) {
	size_t dim_x;
	size_t dim_y;
	size_t dim_z;
	int offset;
	int device_id;
	int nRuns;

	// Check!
	char * pEnd;
	if (argc==7) {
		dim_x        = strtol(argv[1],&pEnd,10); // this with CONV_SIZE gives signal size
		dim_y        = strtol(argv[2],&pEnd,10);
		dim_z        = strtol(argv[3],&pEnd,10);
		offset       = strtol(argv[4],&pEnd,10);
		device_id    = strtol(argv[5],&pEnd,10);
		nRuns        = strtol(argv[6],&pEnd,10);
	}
	else {
		printf("Argument error!\n");
		printf(" 1) dimensions x\n");
		printf(" 2) dimensions y\n");
		printf(" 3) dimensions z\n");
		printf(" 4) offset\n");
		printf(" 5) device id\n");
		printf(" 6) number of GPU kernel runs (optional)\n");
        return(1);
	}
	
	if(DEBUG) {
		printf("dim_x:        %zu\n",dim_x);
		printf("dim_y:        %zu\n",dim_y);
		printf("dim_z:        %zu\n",dim_z);
		printf("offset:       %d\n",offset);
		printf("device id:    %d\n",device_id);
		printf("nRuns:        %d\n",nRuns);
	}
	
	
	//----------------> GSL stuff 
	//const gsl_rng_type *rndType;
	//gsl_rng *rnd_handle;
	//gsl_rng_env_setup();
	//long int seed=(long int) time(NULL);
	//rndType = gsl_rng_default;
	//rnd_handle = gsl_rng_alloc (rndType);
	//gsl_rng_set(rnd_handle,seed);
	//----------------> GSL stuff 
	
	size_t input_size = dim_x*dim_y*dim_z;
	size_t MSD_size = 2;
	
	
	if(VERBOSE) printf("Input:%0.3f MB;\n",input_size*4.0/(1024.0*1024.0));
	if(VERBOSE) printf("\t\tWelcome\n");

	float *h_input;
	float *h_MSD;
	size_t h_MSD_nElements;

	h_input		 = (float *)malloc(input_size*sizeof(float));
	h_MSD 		 = (float *)malloc(MSD_size*sizeof(float));
	memset(h_MSD, 0.0, MSD_size*sizeof(float));

	srand(time(NULL));
	Generate_dataset(h_input, dim_x, dim_y, dim_z, offset);
	
	//----------------------------------------------------->
	//------------------------ DEVICE --------------------->
	int deviceCount;
	hipError_t error_id;
	error_id = hipGetDeviceCount(&deviceCount);
	if(error_id != hipSuccess) {
		printf("CUDA ERROR: %s\n", hipGetErrorString(error_id) );
		return(1);
	}
	if(device_id>=deviceCount) {
		printf("Selected device is not available! Device id is %d;\n", device_id);
		return(1);
	}
	if (hipSetDevice(device_id) != hipSuccess) {
		printf("ERROR! unable to set the device with id %d.\n", device_id);
		return(1);
	}
	
	//---------> Checking memory
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	float free_memory = (float) free_mem/(1024.0*1024.0);
	float memory_required = (input_size*sizeof(float))/(1024.0*1024.0);
	printf("\n");
	printf("Device has %0.3f MB of total memory, which %0.3f MB is available. Memory required %0.3f MB\n", (float) total_mem/(1024.0*1024.0), free_memory ,memory_required);
	if(memory_required>free_memory) {
		printf("\n \n Array is too big for the device! \n \n"); 
		return(1);
	}
	
	//---------> Measurements
	double transfer_in, transfer_out;
	transfer_in=0.0; transfer_out=0.0;
	GpuTimer timer;
	
	//---------> Memory allocation
	float *d_input;
	float *d_MSD;
	size_t *d_MSD_nElements;
	hipMalloc((void **) &d_input,  sizeof(float)*input_size);
	hipMalloc((void **) &d_MSD, sizeof(float)*MSD_RESULTS_SIZE);
	hipMalloc((void **) &d_MSD_nElements, sizeof(size_t));
	
	//---------> Copy data to the device
	printf("Data transfer to the device memory...: \t");
	timer.Start();
	hipMemcpy(d_input, h_input, input_size*sizeof(float), hipMemcpyHostToDevice);
	timer.Stop();
	transfer_in+=timer.Elapsed();
	printf("done in %g ms.\n", timer.Elapsed());
	
	//---------> Create MSD plan
	MSD_Error MSD_error;
	bool outlier_rejection = false;
	MSD_Configuration MSD_conf;
	std::vector<size_t> dimensions={dim_z, dim_y, dim_x}; // dimensions of the data. Fastest moving coordinate is at the end.
	MSD_error = MSD_conf.Create_MSD_Plan(dimensions, offset, outlier_rejection, 3.0);
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	if(DEBUG) MSD_conf.PrintDebug();
	
	//---------> Get mean and stdev through library
	timer.Start();
	MSD_error = MSD_GetMeanStdev(d_MSD, d_MSD_nElements, d_input, MSD_conf);
	timer.Stop();
	printf("Calculation of mean and standard deviation took %g ms\n", timer.Elapsed());
	printf("MSD GPU library says it took: %g ms\n", MSD_conf.MSD_time);
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	MSD_conf.PrintInfo();
	
	//---------> Copy data to the host
	printf("Data transfer to the host...: \t");
	timer.Start();
	hipMemcpy( h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy( &h_MSD_nElements, d_MSD_nElements, sizeof(h_MSD_nElements), hipMemcpyDeviceToHost);
	timer.Stop();
	transfer_out+=timer.Elapsed();
	printf("done in %g ms.\n", timer.Elapsed());
	
	printf("\nMSD GPU library outputs one float array (for example d_MSD)\n which contains mean as d_MSD[0] and standard deviation as d_MSD[1].\n Values calculated by MSD GPU library are mean = %f; stdev = %f\n\n", h_MSD[0], h_MSD[1]);
	
	//---------> Feeing allocated resources
	hipFree(d_input);
	hipFree(d_MSD);
	hipFree(d_MSD_nElements);
	MSD_conf.Destroy_MSD_Plan();
	//------------------------ DEVICE ---------------------<
	//-----------------------------------------------------<
	
	if (CHECK){
		double signal_mean, signal_sd, merror, sderror;
		MSD_Kahan(h_input, dim_x, dim_y, dim_z, offset, &signal_mean, &signal_sd);
		merror  = sqrt((signal_mean-h_MSD[0])*(signal_mean-h_MSD[0]));
		sderror = sqrt((signal_sd-h_MSD[1])*(signal_sd-h_MSD[1]));
		if(merror<1e-3 && sderror<1e-2) printf("     Test:\033[1;32mPASSED\033[0m\n");
		else printf("     Test:\033[1;31mFAILED\033[0m\n     Difference Kahan-GPU Mean:%e; Standard deviation:%e;\n", merror, sderror);
		
		printf("GPU results: Mean: %e, Standard deviation: %e; Number of elements:%zu;\n", h_MSD[0], h_MSD[1], h_MSD_nElements);
		printf("MSD_kahan:   Mean: %e, Standard deviation: %e;\n",signal_mean, signal_sd);
		printf("Difference Kahan-GPU Mean:%e; Standard deviation:%e;\n", merror, sderror);
	}
	
	free(h_input);
	free(h_MSD);

	hipDeviceReset();
	
	if (VERBOSE) printf("Finished!\n");

	return (0);
}

